#include "hip/hip_runtime.h"
#include "Common.h"


#define TriangleSize (1<< 5)


__global__ void SierpinskiTriangleKernel(uchar* d_DataOut,uint limit)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y* blockDim.y;

	if (x >= PIXELDIM || y >= PIXELDIM)
		return;

	for (; y >= 0; y--)
	{

		// printing space till 
		// the value of y 
		for (int i = 0; i < y; i++)
		{
			//outFile << " ";
			d_DataOut[x + PIXELDIM *  y] = 0xff; // b
			d_DataOut[x  + PIXELDIM * y + PIXELDIM2] = 0xff; // g
			d_DataOut[x  + PIXELDIM * y + PIXELDIM2 + PIXELDIM2] = 0xff; // r
		

		}
		// printing '*' 
		for (; x + y < PIXELDIM; x++)
		{

			// printing '*' at the appropriate position 
			// is done by the and value of x and y 
			// wherever value is 0 we have printed '*' 
			if (x & y)
			{

				d_DataOut[x + PIXELDIM * y] = 0xff; // b
				d_DataOut[x + PIXELDIM * y + PIXELDIM2] = 0xff; // g
				d_DataOut[x + PIXELDIM * y + PIXELDIM2 + PIXELDIM2] = 0xff; // r
				//outFile << " " << " ";
				//SetData(x, y, 255, data);
				//SetData(x, y, 255, data);
			}
			else
			{
				d_DataOut[x + PIXELDIM * y] = 0x00; // b
				d_DataOut[x + PIXELDIM * y + PIXELDIM2] = 0x00; // g
				d_DataOut[x + PIXELDIM * y + PIXELDIM2 + PIXELDIM2] = 0x00; // r
				//SetData(x, y, 0, data);
				//outFile << "* "; 
				
			}

		}

		//outFile << endl; 
	}


}



#define STriangleDefault
void STriangle::TriangleGPU(uchar** gpuOutput)
{
#ifdef STriangleDefault

	dim3 Block(TriangleSize, TriangleSize, 1);
	dim3 Grid(ceil(((float)PIXELDIM) / TriangleSize), ceil(((float)PIXELDIM) / TriangleSize), 1);


	checkCudaErrors(hipMalloc((void **)&ptr1, PIXELDIM3 * sizeof(uchar)));
	SierpinskiTriangleKernel << <Grid, Block >> > (ptr1,PIXELDIM);

	hipDeviceSynchronize();

	*gpuOutput = (uchar *)malloc(PIXELDIM3 * sizeof(uchar));
	checkCudaErrors(hipMemcpy(*gpuOutput, ptr1, PIXELDIM3 * sizeof(uchar), hipMemcpyDeviceToHost));


#endif
}
void STriangle::ClearMemory(uchar**data)
{
#ifdef STriangleDefault

	hipFree(ptr1);
	free(*data);


#endif
}