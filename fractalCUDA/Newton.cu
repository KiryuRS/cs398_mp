#include "hip/hip_runtime.h"
#include "Common.h"
#include <thrust/complex.h> // mimics std::complex (more natural looking code)
#include <hip/hip_complex.h>

//__device__ hipFloatComplex roots[3] =
//{
//  hipFloatComplex{ 1.0f, 0.0f },
//  hipFloatComplex{ -0.5f,  sqrtf(3.0f) / 2.0f },
//  hipFloatComplex{ -0.5f, -sqrtf(3.0f) / 2.0f }
//};

__device__ thrust::complex<float> Fz(const thrust::complex<float>& z)
{
  return z * z*z - thrust::complex<float>(1.0f, 0.0f);
}

__device__ thrust::complex<float> dFz(const thrust::complex<float>& z)
{
  return thrust::complex<float>(3.0f, 0.0f) * (z*z);
}

__device__ hipFloatComplex cuFz(const hipFloatComplex& z)
{
  return hipCsubf(hipCmulf(hipCmulf(z, z), z), hipFloatComplex{ 1.0f, 0.0f });
}

__device__ hipFloatComplex cudFz(const hipFloatComplex& z)
{
  return hipCmulf(hipFloatComplex{ 3.0f, 0.0f }, hipCmulf(z, z));
}

__device__ void SetDataGPU(int x, int y, uchar* data, int color)
{
  size_t index = y * PIXELDIM + x;
  if (index < PIXELDIM2)
  {
    switch (color)
    {
    case 0:
      data[index + PIXELDIM2 + PIXELDIM2] = 0xff; // r
      data[index + PIXELDIM2] = 0x00;
      data[index] = 0x00;
      break;
    case 1:
      data[index + PIXELDIM2 + PIXELDIM2] = 0x00;
      data[index + PIXELDIM2] = 0xff; // g
      data[index] = 0x00;
      break;
    case 2:
      data[index + PIXELDIM2 + PIXELDIM2] = 0x00;
      data[index + PIXELDIM2] = 0x00;
      data[index] = 0xff; // b
      break;
    }
  }
}

__global__ void NewtonGPUCalc(uchar *d_DataOut)
{
  // __shared__ thrust::complex<float> roots[3];

  int tx = threadIdx.x + blockIdx.x * blockDim.x;
  int ty = threadIdx.y + blockIdx.y * blockDim.y;

  if (tx >= PIXELDIM || ty >= PIXELDIM)
    return;

  float zx = (float)tx * 2.0f / (PIXELDIM - 1) + -1.0f;
  float zy = (float)ty * 2.0f / (PIXELDIM - 1) + -1.0f;

  //// Mapped coordinates
  //thrust::complex<float> z{ zx, zy };

  ////// Roots of polynomials
  //thrust::complex<float> roots[3] =
  //{
  //  thrust::complex<float>{ 1.0f, 0.0f },
  //  thrust::complex<float>{ -0.5f,  sqrtf(3.0f) / 2.0f },
  //  thrust::complex<float>{ -0.5f, -sqrtf(3.0f) / 2.0f }
  //};

    // Mapped coordinates
  hipFloatComplex z{ zx, zy };

  //// Roots of polynomials
  hipFloatComplex roots[3] =
  {
    hipFloatComplex{ 1.0f, 0.0f },
    hipFloatComplex{ -0.5f,  sqrtf(3.0f) / 2.0f },
    hipFloatComplex{ -0.5f, -sqrtf(3.0f) / 2.0f }
  };

  int iteration = 0;
  bool done = false;
  while (iteration < MAX_ITERATIONS && !done)
  {
    z = hipCsubf(z, hipCdivf(cuFz(z), cudFz(z)));

    for (int i = 0; i < 3; ++i)
    {
      // thrust::complex<float> diff = (z - roots[i]);
      hipFloatComplex diff = hipCsubf(z, roots[i]);

      if (std::fabsf(diff.x) < EPSILON && std::fabsf(diff.y) < EPSILON)
      {
        SetDataGPU(tx, ty, d_DataOut, i);
        done = true;
        break;
      }
    }
    ++iteration;
    __syncthreads();
  }
}

void NewtonGPU(uchar* data)
{
  dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 DimGrid((uint)ceil(((float)PIXELDIM) / BLOCK_SIZE), (uint)ceil(((float)PIXELDIM) / BLOCK_SIZE), 1);

  // Allocate memory
  uchar* data_gpu;
  checkCudaErrors(hipMalloc(&data_gpu, PIXELDIM3 * sizeof(uchar)));
  checkCudaErrors(hipMemcpy(data_gpu, data, PIXELDIM3 * sizeof(uchar), hipMemcpyHostToDevice));

  NewtonGPUCalc<<<DimGrid, DimBlock>>>(data_gpu);
  checkCudaErrors(hipGetLastError());

  hipDeviceSynchronize();

  checkCudaErrors(hipMemcpy(data, data_gpu, PIXELDIM3 * sizeof(uchar), hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(data_gpu));
}