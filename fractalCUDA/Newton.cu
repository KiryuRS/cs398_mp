#include "hip/hip_runtime.h"
/******************************************************************************/
/*!
@file   Newton.cu
@par    Purpose: Implementation of Newton CUDA kernel
@par    Language: C++
@par    Platform: Visual Studio 2015, Windows 10 64-bit
@author Alvin
@par    Email: alvin.tan\@digipen.edu
@date   07/12/2018
*/
/******************************************************************************/
#include "Common.h"
#include <thrust/complex.h> // mimics std::complex (more natural looking code)
#include <hip/hip_complex.h>

__device__ thrust::complex<float> Fz(const thrust::complex<float>& z)
{
	return z * z*z - thrust::complex<float>(1.0f, 0.0f);
}

__device__ thrust::complex<float> dFz(const thrust::complex<float>& z)
{
	return thrust::complex<float>(3.0f, 0.0f) * (z*z);
}

__device__ hipFloatComplex cuFz(const hipFloatComplex& z)
{
	return hipCsubf(hipCmulf(hipCmulf(z, z), z), hipFloatComplex{ 1.0f, 0.0f });
}

__device__ hipFloatComplex cudFz(const hipFloatComplex& z)
{
	return hipCmulf(hipFloatComplex{ 3.0f, 0.0f }, hipCmulf(z, z));
}

__device__ void SetDataGPU(int x, int y, uchar* data, int color)
{
	size_t index = y * PIXELDIM + x;
	if (index < PIXELDIM2)
	{
		switch (color)
		{
		case 0:
			data[index + PIXELDIM2 + PIXELDIM2] = 0xff; // r
			data[index + PIXELDIM2] = 0x00;
			data[index] = 0x00;
			break;
		case 1:
			data[index + PIXELDIM2 + PIXELDIM2] = 0x00;
			data[index + PIXELDIM2] = 0xff; // g
			data[index] = 0x00;
			break;
		case 2:
			data[index + PIXELDIM2 + PIXELDIM2] = 0x00;
			data[index + PIXELDIM2] = 0x00;
			data[index] = 0xff; // b
			break;
		}
	}
}

#if defined CUCOMPLEX_VERSION
__global__ void NewtonGPUCalc(uchar *d_DataOut)
{
	// __shared__ thrust::complex<float> roots[3];

	//__shared__ int x_shared[BLOCK_SIZE];
	//__shared__ int y_shared[BLOCK_SIZE];
	//__shared__ int c_shared[BLOCK_SIZE];

	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;

	if (tx >= PIXELDIM || ty >= PIXELDIM)
		return;

	float zx = (float)tx * 2.0f / (PIXELDIM - 1) + -1.0f;
	float zy = (float)ty * 2.0f / (PIXELDIM - 1) + -1.0f;

	//// Mapped coordinates
	//thrust::complex<float> z{ zx, zy };

	////// Roots of polynomials
	//thrust::complex<float> roots[3] =
	//{
	//  thrust::complex<float>{ 1.0f, 0.0f },
	//  thrust::complex<float>{ -0.5f,  sqrtf(3.0f) / 2.0f },
	//  thrust::complex<float>{ -0.5f, -sqrtf(3.0f) / 2.0f }
	//};

	// Mapped coordinates
	hipFloatComplex z{ zx, zy };

	//// Roots of polynomials
	hipFloatComplex roots[3] =
	{
		hipFloatComplex{ 1.0f, 0.0f },
		hipFloatComplex{ -0.5f,  sqrtf(3.0f) / 2.0f },
		hipFloatComplex{ -0.5f, -sqrtf(3.0f) / 2.0f }
	};

	int iteration = 0;
	bool done = false;
	while (iteration < MAX_ITERATIONS && !done)
	{
		z = hipCsubf(z, hipCdivf(cuFz(z), cudFz(z)));

		for (int i = 0; i < 3; ++i)
		{
			// thrust::complex<float> diff = (z - roots[i]);
			hipFloatComplex diff = hipCsubf(z, roots[i]);

			if (std::fabsf(diff.x) < EPSILON && std::fabsf(diff.y) < EPSILON)
			{
				SetDataGPU(tx, ty, d_DataOut, i);
				//x_shared[tx % BLOCK_SIZE] = tx;
				//y_shared[tx % BLOCK_SIZE] = ty;
				//c_shared[tx % BLOCK_SIZE] = i;
				done = true;
				break;
			}
		}
		++iteration;
		__syncthreads();
	}
}
#elif defined THRUST_VERSION
__global__ void NewtonGPUCalc(uchar *d_DataOut)
{
	// __shared__ thrust::complex<float> roots[3];

	//__shared__ int x_shared[BLOCK_SIZE];
	//__shared__ int y_shared[BLOCK_SIZE];
	//__shared__ int c_shared[BLOCK_SIZE];

	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int ty = threadIdx.y + blockIdx.y * blockDim.y;

	if (tx >= PIXELDIM || ty >= PIXELDIM)
		return;

	float zx = (float)tx * 2.0f / (PIXELDIM - 1) + -1.0f;
	float zy = (float)ty * 2.0f / (PIXELDIM - 1) + -1.0f;

	// Mapped coordinates
	thrust::complex<float> z{ zx, zy };

	//// Roots of polynomials
	thrust::complex<float> roots[3] =
	{
	  thrust::complex<float>{ 1.0f, 0.0f },
	  thrust::complex<float>{ -0.5f,  sqrtf(3.0f) / 2.0f },
	  thrust::complex<float>{ -0.5f, -sqrtf(3.0f) / 2.0f }
	};

	int iteration = 0;
	bool done = false;
	while (iteration < MAX_ITERATIONS && !done)
	{
		z -= Fz(z) / dFz(z);
		//z = hipCsubf(z, hipCdivf(cuFz(z), cudFz(z)));

		for (int i = 0; i < 3; ++i)
		{
			thrust::complex<float> diff = (z - roots[i]);
			//hipFloatComplex diff = hipCsubf(z, roots[i]);

			if (std::fabsf(diff.real()) < EPSILON && std::fabsf(diff.imag()) < EPSILON)
			{
				SetDataGPU(tx, ty, d_DataOut, i);
				done = true;
				break;
			}
		}
		++iteration;
		__syncthreads();
	}
}
#endif

void NewtonGPU(uchar* data)
{
  dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  dim3 DimGrid((uint)ceil(((float)PIXELDIM) / BLOCK_SIZE), (uint)ceil(((float)PIXELDIM) / BLOCK_SIZE), 1);

  uchar* data_gpu;
  checkCudaErrors(hipMalloc(&data_gpu, PIXELDIM3 * sizeof(uchar)));
  checkCudaErrors(hipMemcpy(data_gpu, data, PIXELDIM3 * sizeof(uchar), hipMemcpyHostToDevice));
  checkCudaErrors(hipGetLastError());

  NewtonGPUCalc<<<DimGrid, DimBlock>>>(data_gpu);
  checkCudaErrors(hipGetLastError());

  hipDeviceSynchronize();

  checkCudaErrors(hipMemcpy(data, data_gpu, PIXELDIM3 * sizeof(uchar), hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(data_gpu));
}